#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include "cudafile2.cuh"
#include <stdio.h>
#include <iostream>
#include ""

using namespace std;

#define N 10

void fillalldata(int ** data[N][N]) {
	
	for (int i = 0; i < N;i++) {
		for (int j = 0; j < N; j++) {
			**data[i][j] = j * 4;
		 }
	}
}

__global__ void AddMatrix(int a[][N],int b[][N],int c[][N]) {

	int i = threadIdx.x;
	int j = threadIdx.y;
	c[i][j] = a[i][j] + b[i][j];
}


void PrintEndValue(int  a[N][N], int  b[N][N], int  c[N][N]) {
	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < N;j++)
		{
			printf("%d %d %d", a[i][j], b[i][j], c[i][j]);
		}
		printf("\n");
	}
}



void RunCudaFile2()
{
	

}