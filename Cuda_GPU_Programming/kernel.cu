#include "hip/hip_runtime.h"
﻿

#include "hip/hip_runtime.h"
#include ""
#include <stdarg.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include "cudafile2.cuh"

/**
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>
#include <hip/device_functions.h>
*/


#define SIZE 1024
#define Th 10
#define N 512
#define N1 1024
#define N2 1024


__global__ void Vectoradd(int* arr1, int* arr2, int* arr3, int n)
{

	int i = threadIdx.x;

	if(i < n) {
		arr3[i] = arr2[i] + arr1[i];
	}
}

__device__ int PrintFromDeviceFunction()
{
	printf("Hello World From Device Thread [%d , %d] \n", threadIdx, blockIdx);
	return 10;
}

__global__ void PrintFromGpu()
{
	printf("Hello World From GPU Thread [%d , %d] \n", threadIdx, blockIdx);
	printf("%d",PrintFromDeviceFunction());
}


void Excer1()
{
	int* arr1;
	int *arr2;
	int *arr3;

	hipMallocManaged(&arr1, SIZE * sizeof(int));
	hipMallocManaged(&arr2, SIZE * sizeof(int));
	hipMallocManaged(&arr3, SIZE * sizeof(int));

	for (int i = 0; i < SIZE; i++) {

		arr1[i] = i;
		arr2[i] = i;
		arr3[i] = 0;
	}

	Vectoradd << <1, SIZE >> > (arr1, arr2, arr3, SIZE);

	hipDeviceSynchronize();

	for (int i = 0; i < 1024; i++) {
		printf("C[%d] = %d\n", i, arr3[i]);
	}

	PrintFromGpu << <1, 1 >> > ();

	hipDeviceSynchronize();

	//PrintFromDeviceFunction << <1, 1 >> > ();

	//hipDeviceSynchronize();

	hipFree(arr1);
	hipFree(arr2);
	hipFree(arr3);
}

__global__ void global_device_add(int* a, int* b, int* c)
{
	c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}

void fill_array(int* data)
{
	for (int i = 0; i < N; i++) {
		data[i] = i;
	}
}

void Exer2()
{
	int* arr1; 
	int* arr2; 
	int* arr3;
	int* devicearr1;
	int* devicearr2;
	int* devicearr3;

	arr1 = (int*)malloc(N * sizeof(int));
	arr2 = (int*)malloc(N * sizeof(int));
	arr3 = (int*)malloc(N * sizeof(int));
	fill_array(arr1);
	fill_array(arr2);

	hipMalloc((void**)&devicearr1, N * sizeof(int));
	hipMalloc((void**)&devicearr2, N * sizeof(int));
	hipMalloc((void**)&devicearr3, N * sizeof(int));

	hipMemcpy(devicearr1, arr1, N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(devicearr2, arr2, N * sizeof(int), hipMemcpyHostToDevice);

	global_device_add << <N, 1 >> > (da, db, dc);

	hipMemcpy(arr3, devicearr3, N * sizeof(int), hipMemcpyDeviceToHost);

	for (int i = 0; i < N; i++) {
		printf(" %d %d %d", arr1[i], arr2[i], arr3[i]);
	}

	free(arr1);
	free(arr2);
	free(arr3);

	hipFree(devicearr1);
	hipFree(devicearr2);
	hipFree(devicearr3);
}


__global__ void Device_Add_Thread_Block(int* a, int* b, int* c)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	printf("Info is %d threadIdx.x %d  blockIdx.x %d * blockDim.x \n", threadIdx.x, blockIdx.x, blockDim.x);
	c[index] = a[index] + b[index];
}

void fill_array_two(int* arr)
{
	for (int i = 0;i < N1; i++)
	{
		arr[i] = i;
	}
}


void Thread_Block()
{
	int* arr1;
	int* arr2;
	int* arr3;
	int* devicearr1;
	int* devicearr2;
	int* devicearr3;


	arr1 = (int*)malloc(N1 * sizeof(int));
	arr2 = (int*)malloc(N1 * sizeof(int));
	arr3 = (int*)malloc(N1 * sizeof(int));
	fill_array_two(arr1);
	fill_array_two(arr2);

	hipMalloc((void**)&devicearr1, N1 * sizeof(int));
	hipMalloc((void**)&devicearr2, N1 * sizeof(int));
	hipMalloc((void**)&devicearr3, N1 * sizeof(int));

	hipMemcpy(devicearr1, arr1, N1 * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(devicearr2, arr2, N1 * sizeof(int), hipMemcpyHostToDevice);

	
	int blocks = N1 / 4;

	Device_Add_Thread_Block << <blocks, 4 >> > (devicearr1, devicearr2, devicearr3);

	hipMemcpy(arr3, devicearr3, N1 * sizeof(int), hipMemcpyDeviceToHost);

	for (int i = 0; i < N1; i++)
	{
		printf("%d %d %d", arr1[i], arr2[i], arr3[i]);
	}

	free(arr1);
	free(arr2);
	free(arr3);

	hipFree(devicearr1);
	hipFree(devicearr2);
	hipFree(devicearr3);
}


__global__ void device_mul_fourth(int* a, int* b, int* c)
{
	c[threadIdx.x] = a[threadIdx.x] * b[threadIdx.x] * 100;
}

void Fill_data_Fourth(int* data)
{
	for (int i = 0; i < N2; i++)
	{
		data[i] = i;
	}
}


void RunOnlyThreads()
{
	int* arr1;
	int* arr2;
	int* arr3;
	int* devicearr1;
	int* devicearr2;
	int* devicearr3;

	arr1 = (int*)malloc(N2 * sizeof(int));
	arr2 = (int*)malloc(N2 * sizeof(int));
	arr3 = (int*)malloc(N2 * sizeof(int));
	Fill_data_Fourth(arr1);
	Fill_data_Fourth(arr2);

	hipMalloc((void**)&devicearr1, N2 * sizeof(int));
	hipMalloc((void**)&devicearr2, N2 * sizeof(int));
	hipMalloc((void**)&devicearr3, N2 * sizeof(int));

	hipMemcpy(devicearr1, arr1, N2 * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(devicearr2, arr2, N2 * sizeof(int), hipMemcpyHostToDevice);

	device_mul_fourth<<<1, 1024>>>(da, db, dc);

	hipMemcpy(arr3, devicearr3, N2 * sizeof(int), hipMemcpyDeviceToHost);

	for (int i = 0; i < N2; i++)
	{
		printf(" %d  %d  %d  \n", arr1[i], arr2[i], arr3[i]);
	}

	free(arr1);
	free(arr2);
	free(arr3);

	hipFree(devicearr1);
	hipFree(devicearr2);
	hipFree(devicearr3);

}

__shared__ int timeth;

__global__ void PrintThreadName()
{
	int i = threadIdx.x;

	printf("The Thread Number is %d \n", i);

	__syncthreads();
	timeth += 1;
	printf(" total threads were %d", timeth);
	__syncthreads();
}

__global__ void PrintTimes()
{
	printf(" total threads were %d", timeth);
}



void RunTesting()
{
	PrintThreadName << <1, 1024 >> > ();
	PrintTimes << <1, 1 >> > ();
}


int main() {

	
	Excer1();
	Exer2();
	Thread_Block();
	RunOnlyThreads();
	RunCudaFile2();
	RunTesting();

	return 0;
}


